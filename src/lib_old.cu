#define __USE_GNU

#include <dlfcn.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <iostream>

extern "C" {
void *__libc_dlsym(void *map, const char *name);
}
extern "C" {
void *__libc_dlopen_mode(const char *name, int mode);
}

#define STRINGIFY(x) #x
#define CUDA_SYMBOL_STRING(x) STRINGIFY(x)
#define SYNCP_MESSAGE

typedef void *(*fnDlsym)(void *, const char *);

static void *real_dlsym(void *handle, const char *symbol) {
  static fnDlsym internal_dlsym =
      (fnDlsym)__libc_dlsym(__libc_dlopen_mode("libdl.so.2", RTLD_LAZY), "dlsym");
  return (*internal_dlsym)(handle, symbol);
}

/*
** If the user of this library does not wish to include CUDA specific
*code/headers in the code,
** then all the parameters can be changed and/or simply casted before calling
*the callback.
*/
#define CU_HOOK_GENERATE_INTERCEPT(funcname, params,callback_pre, callback_post, ...)        \
  hipError_t CUDAAPI funcname params {                                         \
    static void* real_func =                                                 \
        (void*)real_dlsym(RTLD_NEXT, CUDA_SYMBOL_STRING(funcname));          \
    hipError_t result = hipSuccess;                                          \
                                                                             \
                                                                      \
    if (callback_pre) {                                         \
      ((hipError_t CUDAAPI(*) params)callback_pre)(__VA_ARGS__);  \
    }   \
    if(!real_func ){std::cout << "error" << std::endl;}                             \
    result = ((hipError_t CUDAAPI(*) params)real_func)(__VA_ARGS__);           \
    if (callback_post && result == hipSuccess) {              \
      ((hipError_t CUDAAPI(*) params)callback_post)(__VA_ARGS__); \
    }                                                                        \
    return (result);                                                         \
  }



/*
 ** Interposed Functions
 */
void *dlsym(void *handle, const char *symbol) {
  // Early out if not a CUDA driver symbol
  if (strncmp(symbol, "cu", 2) != 0) {
    return (real_dlsym(handle, symbol));
  }

  if (strcmp(symbol, CUDA_SYMBOL_STRING(hipMalloc)) == 0) {
    return (void *)(&hipMalloc);
  } else if (strcmp(symbol, CUDA_SYMBOL_STRING(hipFree)) == 0) {
    return (void *)(&hipFree);
  } 
  //else if (strcmp(symbol, CUDA_SYMBOL_STRING(cuCtxGetCurrent)) == 0) {
  //  return (void *)(&cuCtxGetCurrent);
  //} else if (strcmp(symbol, CUDA_SYMBOL_STRING(cuCtxSetCurrent)) == 0) {
  //  return (void *)(&cuCtxSetCurrent);
  //} else if (strcmp(symbol, CUDA_SYMBOL_STRING(cuCtxDestroy)) == 0) {
  //  return (void *)(&cuCtxDestroy);
  //}
  return (real_dlsym(handle, symbol));
}



void cu_malloc_callback(hipDeviceptr_t * dptr, size_t bytesize){
    std::cout << "cu_malloc_callback"<< dptr << " " << bytesize << std::endl;
}
void cu_free_callback(hipDeviceptr_t * dptr){
    std::cout << "cu_free_callback"<< dptr << std::endl;
}

CU_HOOK_GENERATE_INTERCEPT(hipMalloc,
                           (hipDeviceptr_t * dptr, size_t bytesize),nullptr,cu_malloc_callback, dptr,
                           bytesize)

CU_HOOK_GENERATE_INTERCEPT(hipFree, (hipDeviceptr_t dptr),nullptr,cu_free_callback,
                           dptr)


//CU_HOOK_GENERATE_INTERCEPT(cuCtxGetCurrent,
//                           (CUcontext * pctx), pctx)
//CU_HOOK_GENERATE_INTERCEPT(cuCtxSetCurrent,
//                           (CUcontext ctx), ctx)
//CU_HOOK_GENERATE_INTERCEPT(cuCtxDestroy, (CUcontext ctx),
//                           ctx)