#include "hip/hip_runtime.h"

#include <math.h>
#include <stdio.h>
#include <sys/time.h>

const int Row = 2048;
const int Col = 2048;

__global__ void matrix_mul_gpu(int *M, int *N, int *P, int width) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;

  int sum = 0;
  for (int k = 0; k < width; k++) {
    int a = M[j * width + k];
    int b = N[k * width + i];
    sum += a * b;
  }
  P[j * width + i] = sum;
}

int main() {
  hipError_t cuda_err = hipSuccess;
  printf("func start \n");
  int *A = (int *)malloc(sizeof(int) * Row * Col);
  int *B = (int *)malloc(sizeof(int) * Row * Col);
  int *C = (int *)malloc(sizeof(int) * Row * Col);
  // malloc device memory
  int *d_dataA, *d_dataB, *d_dataC;
  printf("before cudaMalloc()\n");
  cuda_err = hipMalloc((void **)&d_dataA, sizeof(int) * Row * Col);
  // cuda_err = cudaGetLastError();
  if (hipSuccess != cuda_err) {
    fprintf(stderr, "(%s:%s:%d)", __FILE__, __FUNCTION__, __LINE__);
    fprintf(stderr, "%s\n", hipGetErrorString(cuda_err));
    printf("cuda_err is %d\n", cuda_err);
    exit(1);
  }
  printf("after cudaMalloc()\n");
  hipMalloc((void **)&d_dataB, sizeof(int) * Row * Col);
  hipMalloc((void **)&d_dataC, sizeof(int) * Row * Col);
  // set value
  for (int i = 0; i < Row * Col; i++) {
    A[i] = 90;
    B[i] = 10;
  }

  hipMemcpy(d_dataA, A, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
  hipMemcpy(d_dataB, B, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
  dim3 threadPerBlock(16, 16);
  dim3 blockNumber((Col + threadPerBlock.x - 1) / threadPerBlock.x,
                   (Row + threadPerBlock.y - 1) / threadPerBlock.y);
  matrix_mul_gpu<<<blockNumber, threadPerBlock>>>(d_dataA, d_dataB, d_dataC,
                                                  Col);
  hipDeviceSynchronize();
  hipMemcpy(C, d_dataC, sizeof(int) * Row * Col, hipMemcpyDeviceToHost);
  free(A);
  free(B);
  free(C);
  hipFree(d_dataA);
  hipFree(d_dataB);
  hipFree(d_dataC);

  return 0;
}